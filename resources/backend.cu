
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>

// Rading material:
//
// http://users.wfu.edu/choss/CUDA/docs/Lecture%205.pdf
// https://docs.nvidia.com/deeplearning/performance/pdf/GPU-Performance-Background-User-Guide.pdf
// https://developer.nvidia.com/nvidia-visual-profiler

__global__ void add_kernel(double *a, double *b, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

extern "C" void add(double *a, double *b, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  add_kernel<<<gridDim, blockDim>>>(a, b, c, n);
}

__global__ void sub_kernel(double *a, double *b, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] - b[i];
  }
}

extern "C" void sub(double *a, double *b, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  sub_kernel<<<gridDim, blockDim>>>(a, b, c, n);
}

__global__ void mul_kernel(double *a, double *b, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] * b[i];
  }
}

extern "C" void mul(double *a, double *b, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  mul_kernel<<<gridDim, blockDim>>>(a, b, c, n);
}

__global__ void div_kernel(double *a, double *b, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] / b[i];
  }
}

// not named "div" because it exists already
extern "C" void division(double *a, double *b, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  div_kernel<<<gridDim, blockDim>>>(a, b, c, n);
}

__global__ void sqrt_kernel(double *a, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = sqrtf(a[i]);
  }
}

extern "C" void rusty_sqrt(double *a, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  sqrt_kernel<<<gridDim, blockDim>>>(a, c, n);
}

__global__ void log_kernel(double *a, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = log2f(a[i]);
  }
}

extern "C" void rusty_log(double *a, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  log_kernel<<<gridDim, blockDim>>>(a, c, n);
}

__global__ void relu_kernel(double *a, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = fmaxf(a[i], 0.0);
  }
}

extern "C" void relu(double *a, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  relu_kernel<<<gridDim, blockDim>>>(a, c, n);
}

__global__ void sigmoid_kernel(double *a, double *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = (1.0 / (1.0 + expf(-a[i])));
  }
}

extern "C" void sigmoid(double *a, double *c, int n) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  sigmoid_kernel<<<gridDim, blockDim>>>(a, c, n);
}

__global__ void max_kernel(double *a, double *max, int n) {
  extern __shared__ double shared[];

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  double localMax = -DBL_MAX;

  // Local reduction
  for (int i = index; i < n; i += stride) {
    localMax = fmax(localMax, a[i]);
  }

  // Store local max in shared memory
  shared[threadIdx.x] = localMax;
  __syncthreads();

  // Reduction within a block
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      shared[threadIdx.x] = fmax(shared[threadIdx.x], shared[threadIdx.x + s]);
    }
    __syncthreads();
  }

  // First thread in each block writes the result
  if (threadIdx.x == 0) {
    max[blockIdx.x] = shared[0];
  }
}

extern "C" void rusty_max(double *a, double *c, int n) {
  const int blockSize = 256;
  const int numBlocks = (n + blockSize - 1) / blockSize;
  dim3 blockDim(blockSize);
  dim3 gridDim(numBlocks);

  double *max;
  hipMalloc(&max, numBlocks * sizeof(double));

  max_kernel<<<gridDim, blockDim, blockSize * sizeof(double)>>>(a, max, n);

  double *result_max = new double[numBlocks];
  hipMemcpy(result_max, max, numBlocks * sizeof(double),
             hipMemcpyDeviceToHost);

  double finalMax = -DBL_MAX;
  for (int i = 0; i < numBlocks; ++i) {
    finalMax = fmax(finalMax, result_max[i]);
  }

  // copying it back to device to be used by other ops
  // FIXME: is it possible to calculate finalMax on device?
  hipMemcpy(c, &finalMax, 1 * sizeof(double), hipMemcpyHostToDevice);
}

__global__ void min_kernel(double *a, double *max, int n) {
  extern __shared__ double shared[];

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  double localMin = DBL_MAX;

  // Local reduction
  for (int i = index; i < n; i += stride) {
    localMin = fmin(localMin, a[i]);
  }

  // Store local max in shared memory
  shared[threadIdx.x] = localMin;
  __syncthreads();

  // Reduction within a block
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      shared[threadIdx.x] = fmin(shared[threadIdx.x], shared[threadIdx.x + s]);
    }
    __syncthreads();
  }

  // First thread in each block writes the result
  if (threadIdx.x == 0) {
    max[blockIdx.x] = shared[0];
  }
}

extern "C" void rusty_min(double *a, double *c, int n) {
  const int blockSize = 256;
  const int numBlocks = (n + blockSize - 1) / blockSize;
  dim3 blockDim(blockSize);
  dim3 gridDim(numBlocks);

  double *min;
  hipMalloc(&min, numBlocks * sizeof(double));

  min_kernel<<<gridDim, blockDim, blockSize * sizeof(double)>>>(a, min, n);

  double *result_min = new double[numBlocks];
  hipMemcpy(result_min, min, numBlocks * sizeof(double),
             hipMemcpyDeviceToHost);

  double finalMin = DBL_MAX;
  for (int i = 0; i < numBlocks; ++i) {
    finalMin = fmin(finalMin, result_min[i]);
  }

  hipMemcpy(c, &finalMin, 1 * sizeof(double), hipMemcpyHostToDevice);
}

// A: M x K, B: K x N, C: M x N
__global__ void matmul_kernel(double *A, double *B, double *C, int M, int K,
                              int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    double sum = 0.0;
    for (int k = 0; k < K; k++) {
      sum += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

extern "C" void matmul(double *a, double *b, double *c, int M, int K, int N) {
  int blockSize = 16;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid((N + blockSize - 1) / blockSize, (M + blockSize - 1) / blockSize,
               1);
  matmul_kernel<<<dimGrid, dimBlock>>>(a, b, c, M, K, N);
}

__global__ void expand_kernel(double *input, double *output, int output_length,
                              int dim_count, size_t *old_shape,
                              size_t *new_shape) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < output_length) {
    size_t idx = 0;
    size_t factor = 1;
    size_t index = i;

    for (int k = dim_count - 1; k >= 0; k--) {
      size_t size_new = new_shape[k];
      size_t size_old = old_shape[k];
      int old_index = 0;

      if (size_old != 1) {
        old_index = i % size_new;
      }

      idx += old_index * factor;
      factor *= size_old;
      i /= size_new;
    }

    output[index] = input[idx];
  }
}

extern "C" void expand(double *input, double *output, int output_length,
                       int dim_count, size_t *old_shape, size_t *new_shape) {
  dim3 blockDim(256);
  dim3 gridDim((output_length + blockDim.x - 1) / blockDim.x);

  expand_kernel<<<gridDim, blockDim>>>(input, output, output_length, dim_count,
                                       old_shape, new_shape);
}

__global__ void pad2d_kernel(double *input, double *output, int input_length,
                             int dim_count, size_t *shape, size_t *new_shape,
                             size_t *padding) {
  extern __shared__ int shared_mem[];
  int *multi_dim_index = shared_mem + threadIdx.x * dim_count;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < input_length) {
    size_t temp_index = i;

    for (int k = dim_count - 1; k >= 0; k--) {
      size_t size = shape[k];
      multi_dim_index[k] = temp_index % size;
      temp_index /= size;
    }

    // bottom and right padding is added in the initialization
    if (dim_count >= 2) {
      multi_dim_index[dim_count - 2] += padding[2]; // top padding
      multi_dim_index[dim_count - 1] += padding[0]; // left padding
    }

    size_t new_index = 0;
    size_t stride = 1;
    for (int k = dim_count - 1; k >= 0; k--) {
      size_t size = new_shape[k];
      size_t index = multi_dim_index[k];

      new_index += index * stride;
      stride *= size;
    }

    output[new_index] = input[i];
  }
}

extern "C" void pad2d(double *input, double *output, int input_length,
                      int dim_count, size_t *shape, size_t *new_shape,
                      size_t *padding) {
  dim3 blockDim(256);
  dim3 gridDim((input_length + blockDim.x - 1) / blockDim.x);

  size_t sharedMemSize = blockDim.x * dim_count * sizeof(int);

  pad2d_kernel<<<gridDim, blockDim, sharedMemSize>>>(
      input, output, input_length, dim_count, shape, new_shape, padding);
}

__global__ void permute_kernel(double *input, double *output, int input_length,
                               int dim_count, size_t *shape, size_t *new_shape,
                               size_t *dims) {
  extern __shared__ int shared_mem[];
  int *multi_dim_index = shared_mem + threadIdx.x * dim_count;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < input_length) {
    size_t temp_index = i;

    for (int k = dim_count - 1; k >= 0; k--) {
      size_t size = shape[k];
      multi_dim_index[k] = temp_index % size;
      temp_index /= size;
    }

    size_t new_index = 0;
    size_t stride = 1;
    for (int k = dim_count - 1; k >= 0; k--) {
      size_t size = new_shape[k];
      size_t index = multi_dim_index[dims[k]];

      new_index += index * stride;
      stride *= size;
    }

    output[new_index] = input[i];
  }
}

extern "C" void permute(double *input, double *output, int input_length,
                        int dim_count, size_t *shape, size_t *new_shape,
                        size_t *dims) {
  dim3 blockDim(256);
  dim3 gridDim((input_length + blockDim.x - 1) / blockDim.x);

  size_t sharedMemSize = blockDim.x * dim_count * sizeof(int);

  permute_kernel<<<gridDim, blockDim, sharedMemSize>>>(
      input, output, input_length, dim_count, shape, new_shape, dims);
}

__global__ void sum_kernel(double *input, int n) {
  extern __shared__ double shared[];

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  double localMax = -DBL_MAX;

  // Local reduction
  for (int i = index; i < n; i += stride) {
    // TODO
    // implement algorithm used in cpu backend
    // and store in shared memory
  }
  __syncthreads();

  // Reduction within a block
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      // TODO
      // implement algorithm used in cpu backend
      // and store in shared memory
    }
    __syncthreads();
  }

  // First thread in each block writes the results
  if (threadIdx.x == 0) {
    // TODO
    // write to output
  }
}

extern "C" void sum(double *input, double *output, int n, size_t *input_shape,
                    size_t *target_dims) {
  dim3 blockDim(256);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

  sum_kernel<<<gridDim, blockDim>>>(input, n);
}
